#include <hip/hip_runtime.h>

__global__
void rgb_to_gray_kernel(
        const unsigned char* r,
        const unsigned char* g,
        const unsigned char* b,
        unsigned char* gray,
        int width,
        int height
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = width * height;

    if (idx < total) {
        float rf = static_cast<float>(r[idx]);
        float gf = static_cast<float>(g[idx]);
        float bf = static_cast<float>(b[idx]);
        float gray_val = 0.299f * rf + 0.587f * gf + 0.114f * bf;
        gray[idx] = static_cast<unsigned char>(gray_val);
    }
}

extern "C"
void ConvertToGrayCUDA(
        const unsigned char* r,
        const unsigned char* g,
        const unsigned char* b,
        unsigned char* out,
        int width,
        int height
) {
    size_t num_pixels = width * height;
    size_t bytes = num_pixels * sizeof(unsigned char);

    unsigned char *d_r, *d_g, *d_b, *d_gray;

    hipMalloc(&d_r, bytes);
    hipMalloc(&d_g, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_gray, bytes);

    hipMemcpy(d_r, r, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_g, g, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, bytes, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (num_pixels + blockSize - 1) / blockSize;

    rgb_to_gray_kernel<<<numBlocks, blockSize>>>(d_r, d_g, d_b, d_gray, width, height);
    hipDeviceSynchronize();

    hipMemcpy(out, d_gray, bytes, hipMemcpyDeviceToHost);

    hipFree(d_r);
    hipFree(d_g);
    hipFree(d_b);
    hipFree(d_gray);
}